#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void vectorAdd(float *, float *, float *, int);

__global__
void vectorAdd(float *A, float *B, float *C, int n)
{
    // CUDA kernel definition
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n)
    {
        C[i] = A[i] + B[i];
    }
    return;
}

void vecAdd(float *h_A, float *h_B, float *h_C, int n)
{
    // host program
    int size = n * sizeof(float);
    float *d_A = NULL, *d_B = NULL, *d_C = NULL;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    err = hipMalloc((void **) &d_A, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **) &d_B, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **) &d_C, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);
    err = hipGetLastError();
    // device function (CUDA kernel) called from host does not have return type
    // CUDA runtime functions (execute in host side) can have return type

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    // Verify that the result vector is correct
    for (int i = 0; i < n; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");

    return;
}

int main()
{
    int n;
    float *h_A, *h_B, *h_C;
    int i;
    printf("Enter the number of elements: ");
    scanf("%d", &n);

    h_A = (float *) malloc(sizeof(float) * n);
    h_B = (float *) malloc(sizeof(float) * n);
    h_C = (float *) malloc(sizeof(float) * n);

    srand(time(0));
    for (i = 0; i < n; ++i)
    {
        h_A[i] = rand();
        h_B[i] = rand();
    }

    vecAdd(h_A, h_B, h_C, n);
    return 0;
}