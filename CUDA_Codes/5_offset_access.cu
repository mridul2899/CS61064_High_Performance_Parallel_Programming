#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void offset_access(float *a, int s, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid + s < n)
    {
        a[tid + s] = a[tid + s] + 1;
    }
}

__global__ void strided_access(float *a, int s, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid * s < n)
    {
        a[tid * s] = a[tid * s] + 1;
    }
}

int main()
{
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipError_t err = hipSuccess;
    int nMB = 128;
    float ms;
    int blockSize = 1024;
    int n = nMB * 1024 * 1024 / sizeof(float);
    float *d_a;
    err = hipMalloc(&d_a, n * sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Memory not allocated (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    int i;
    for (i = 0; i <= 32; ++i)
    {
        err = hipMemset(d_a, 0.0, n * sizeof(float));
        if (err != hipSuccess)
        {
            fprintf(stderr, "Data not written (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        hipEventRecord(startEvent);
        strided_access<<<n / blockSize, blockSize>>>(d_a, i, n);
        hipEventRecord(stopEvent);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&ms, startEvent, stopEvent);
        printf("%d, %f\n", i, ms);
    }
    printf("Just checking\n");
}